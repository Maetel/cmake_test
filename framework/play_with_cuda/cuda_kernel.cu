#include "hip/hip_runtime.h"
//this is only for CMake tutorial

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

__global__ void cuda_hello() {
	printf("Hello World from GPU!\n");
	//std::cout << "what" << std::endl;	//nvcc compile error!
}

int main() {
    cuda_hello << <1, 1 >> > ();
	//printf("hi");

	std::cout << "what" << std::endl;

	return 0;
}